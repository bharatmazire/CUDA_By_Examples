#include "hip/hip_runtime.h"
/*
	CUDA program.
	VectorAddition.cu
	Date : 28/08/2020
*/

// header
#include<iostream>
#include"../../../Include/book.h"

#define N 10

__global__ void add(int *a, int *b, int *c)
{
	// declaration of variables
	int tid;		// for thread id

	// code
	tid = threadIdx.x;

	if(tid < N)
	{
		c[tid] = a[tid] + b[tid];
	}
}

int main(void)
{
	// declaration of variables
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	// code

	// allocate the memory on GPU
	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

	// fill the array a and b
	for(int i = 0 ; i < N ; i++)
	{
		a[i] = i;
		b[i] = i * i;
	}
	
	// copy the arrays a and b to the GPU
	HANDLE_ERROR( hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice) );

	add<<<1,N>>>(dev_a,dev_b,dev_c);

	// copy c array from GPU to host
	HANDLE_ERROR( hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost) );

	for(int i = 0 ; i < N; i++)
	{
		printf("%d + %d = %d \n",a[i], b[i], c[i]);
	}

	// free the device memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
 	
	return(0);
}
