#include "hip/hip_runtime.h"
/*
	CUDA program.
	VectorAddition.cu
	Date : 28/08/2020

	for use of blocks and thread both
	when we have limited sets of threads and blocks, but still want more parallism
	we can do following kernel code
*/

// header
#include<iostream>
#include"../../../Include/book.h"

#define N 10000

__global__ void add(int *a, int *b, int *c)
{
	// declaration of variables
	int tid;		// for thread id

	// code
	tid = threadIdx.x + blockIdx.x * blockDim.x;

	//if(tid < N)
	while( tid < N )								// iterate till end
	{
		c[tid] 	= a[tid] + b[tid];
		tid 	= tid + blockDim.x * gridDim.x;		// so each thread will get its own uniqe next index till they cross limit
	}
}

int main(void)
{
	// declaration of variables
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	// code

	// allocate the memory on GPU
	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

	// fill the array a and b
	for(int i = 0 ; i < N ; i++)
	{
		a[i] = i;
		b[i] = i * i;
	}
	
	// copy the arrays a and b to the GPU
	HANDLE_ERROR( hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice) );


	int iNumberOfThreads = 256;

	add<<<iNumberOfThreads , iNumberOfThreads >>>(dev_a,dev_b,dev_c);
	

	// copy c array from GPU to host
	HANDLE_ERROR( hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost) );

	for(int i = 0 ; i < N; i++)
	{
		printf("%d + %d = %d \n",a[i], b[i], c[i]);
	}

	// free the device memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
 	
	return(0);
}
