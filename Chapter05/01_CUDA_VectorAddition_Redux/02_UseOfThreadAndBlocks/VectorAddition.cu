#include "hip/hip_runtime.h"
/*
	CUDA program.
	VectorAddition.cu
	Date : 28/08/2020

	for use of blocks and thread both
*/

// header
#include<iostream>
#include"../../../Include/book.h"

#define N 1000

__global__ void add(int *a, int *b, int *c)
{
	// declaration of variables
	int tid;		// for thread id

	// code
	tid = threadIdx.x + blockIdx.x * blockDim.x;

	if(tid < N)
	{
		c[tid] = a[tid] + b[tid];
	}
}

int main(void)
{
	// declaration of variables
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	// code

	// allocate the memory on GPU
	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

	// fill the array a and b
	for(int i = 0 ; i < N ; i++)
	{
		a[i] = i;
		b[i] = i * i;
	}
	
	// copy the arrays a and b to the GPU
	HANDLE_ERROR( hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice) );
	HANDLE_ERROR( hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice) );


	int iNumberOfThreads = 256;
	add<<< (N + (iNumberOfThreads-1))/ iNumberOfThreads , iNumberOfThreads >>>(dev_a,dev_b,dev_c);
	// Why (N + (iNumberOfThreads-1))/ iNumberOfThreads ? - In any case N == 0 or N == 1, we might need to use ceil() function to get roundup value.
	// so intead of that use this approach, which will surely give us the reaquired number

	// copy c array from GPU to host
	HANDLE_ERROR( hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost) );

	for(int i = 0 ; i < N; i++)
	{
		printf("%d + %d = %d \n",a[i], b[i], c[i]);
	}

	// free the device memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
 	
	return(0);
}
