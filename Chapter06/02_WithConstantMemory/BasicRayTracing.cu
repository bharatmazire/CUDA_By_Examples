#include "hip/hip_runtime.h"
#include <stdio.h>
#include<math.h>
#include"book.h"
#include"hip/hip_runtime.h"

#define INF 2e10f
#define rnd( x ) (x * rand() / RAND_MAX)
#define SPHERES 20

struct Sphere 
{
	float r,b,g;
	float radius;
	float x,y,z;

	__device__ float hit( float ox, float oy, float *n ) 
	{
		float dx = ox - x;
		float dy = oy - y;
		if (dx*dx + dy*dy < radius*radius) 
		{
			float dz = sqrtf( radius*radius - dx*dx - dy*dy );
			*n = dz / sqrtf( radius * radius );
			return dz + z;
		}
		return -INF;
	}
};


__constant__ Sphere s[SPHERES];

unsigned char* dev_bitmap;

__global__ void kernel(unsigned char* ptr, int DIM)
{
	// map from threadIdx/BlockIdx to pixel position
	int x =  threadIdx.x + blockIdx.x * blockDim.x;
	int y =  threadIdx.y + blockIdx.y * blockDim.y;

	int offset = x + y * blockDim.x * gridDim.x;

	// now calculate the value at that position
	float ox = (x - DIM/2);
	float oy = (y - DIM/2);
	
	float r=0, g=0, b=0;
	float maxz = -INF;

	for(int i = 0 ; i < SPHERES ; i++) 
	{
		float n;
		float t = s[i].hit( ox, oy, &n );
		if (t > maxz) 
		{
			float fscale = n;
			r = s[i].r * fscale;
			g = s[i].g * fscale;
			b = s[i].b * fscale;
		}
	}

	ptr[offset * 4 + 0] = (int)(r * 255);
	ptr[offset * 4 + 1] = (int)(g * 255);
	ptr[offset * 4 + 2] = (int)(b * 255);
	ptr[offset * 4 + 3] = 255;
}

void InitializeGPU(int DIM, unsigned char* CheckImage)
{
	hipEvent_t start, stop;

	HANDLE_ERROR( hipEventCreate( &start ) );
	HANDLE_ERROR( hipEventCreate( &stop ) );

	HANDLE_ERROR( hipEventRecord( start, 0 ) );

	HANDLE_ERROR(hipMalloc((void**)&dev_bitmap, DIM * DIM * 4));				// allocate memory for bitmap
	HANDLE_ERROR( hipMalloc( (void**)&s,sizeof(Sphere) * SPHERES ) );			// allocate memory for spheres

	Sphere *temp_s = (Sphere*)malloc( sizeof(Sphere) * SPHERES );

	for (int i=0; i<SPHERES; i++) 
	{
		temp_s[i].r 		= rnd( 1.0f );
		temp_s[i].g 		= rnd( 1.0f );
		temp_s[i].b 		= rnd( 1.0f );
		temp_s[i].x 		= rnd( 1000.0f ) - 500;
		temp_s[i].y 		= rnd( 1000.0f ) - 500;
		temp_s[i].z 		= rnd( 1000.0f ) - 500;
		temp_s[i].radius 	= rnd( 100.0f ) + 20;
 	}

	//HANDLE_ERROR( hipMemcpy( s, temp_s, sizeof(Sphere) * SPHERES, hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( s), temp_s, sizeof(Sphere) * SPHERES) );							// for constant memory
	
	free( temp_s );

	dim3 grids(DIM/16, DIM/16);
	dim3 threads(16,16);

	kernel <<<grids, threads>>> (dev_bitmap, DIM);

	HANDLE_ERROR( hipMemcpy(CheckImage, dev_bitmap, (DIM * DIM * 4), hipMemcpyDeviceToHost) );

	hipFree(dev_bitmap);
	hipFree(s);
}
