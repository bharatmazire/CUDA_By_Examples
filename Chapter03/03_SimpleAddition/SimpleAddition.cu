#include "hip/hip_runtime.h"
/*
	CUDA program.
	SimpleAddition.cu
	Date : 25/08/2020
*/

// header
#include<iostream>
#include"../../Include/book.h"


__global__ void add(int a, int b, int *c)
{
	// code
	*c = a + b;
}

int main(void)
{
	// declaration of variables
	int c;
	int *dev_c;

	// code
	// HANDLE_ERROR : macro from book.h, for error check
	HANDLE_ERROR( hipMalloc((void**)&dev_c, sizeof(int)) );	// hipMalloc() to allocate the memory on GPU / Device
	// The first argument is a pointer to the pointer you want to hold the address of the newly allocated memory, 
	// and the second parameter is the size of the allocation you want to make
	


	add<<<1,1>>>(2,7,dev_c);			// call kernel with function parameters in '(...)'

	HANDLE_ERROR( hipMemcpy(&c,dev_c, sizeof(int), hipMemcpyDeviceToHost) );		// hipMemcpy() to copy memory from(and to) GPU. hipMemcpyDeviceToHost : Device to Host => GPU to CPU

	printf("\n\t 2 + 7 is : %d \n\n",c);

	hipFree(dev_c);				// free the cuda memory
	
	return(0);
}
